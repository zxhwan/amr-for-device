#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include "typedef.h"
#include "enc.h"
#include <time.h>




static const char* pass = "pass";
static const char* fail = "fail";

#define AMR_MAGIC_NUMBER "#!AMR-WB\n"

static const char* name[] = { "Word8", "UWord8", "Word16", "Word32", "Float32",
"Float64" };


__device__ static const unsigned long size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};

__global__ void amrenc(int *c) {
	int i, j;
	int dtx = 0;
	*c = dtx + 4;
	enc_interface_State enstate;
	Encoder_Interface_init(&enstate, dtx);
	//dec_interface_State destate;
//	Decoder_Interface_init(&destate);
	//srand(0);
		//int req_mode = 7;
	short speech[160];
	for (j = 0; j < 160; j++) {
		speech[j] = 48;//(short)rand();
	}
	//fwrite(speech, sizeof(short int), 160, pcm_orig);
	unsigned char serial_data[32];
	int byte_counter = Encoder_Interface_Encode(&enstate, (Mode)7, speech, serial_data, 0);
		printf("the result is %d,%s\n",*c, speech);
		//fwrite(serial_data, sizeof(char), byte_counter, amrnb);
		/*printf("test times: %d, bytes: %d\n", i, byte_counter);*/
		//int dec_mode = (serial_data[0] >> 3) & 0x000F;
		//int read_size = block_size[dec_mode];
		//Decoder_Interface_Decode(&destate, serial_data, speech, 0);
		//fwrite(speech, sizeof(short int), 160, pcm_back);
		/*printf("test times: %d, bytes: %d\n", i, read_size);*/

}
int main() {
	int i, j;
	clock_t start, finish;

	double  duration;
	for (i = 0; i < 6; i++) {
		const char* result = (size[i][0] == size[i][1] ? pass : fail);
		printf("%s size: %lu, %s\n", name[i], size[i][0], result);
		if (result == fail) {
			exit(1);
		}

	//FILE* pcm_orig = fopen("cclnb.orig", "wb");
	//FILE* amrnb = fopen("cclnb.amrnb", "wb");
	//FILE* pcm_back = fopen("cclnb.back", "wb");
	//if (NULL == pcm_orig) exit(1);
	//if (NULL == amrnb) exit(1);
	//if (NULL == pcm_back) exit(1);
	//fwrite(AMR_MAGIC_NUMBER, sizeof(char), strlen(AMR_MAGIC_NUMBER), amrnb);
	}
	enc_interface_State *host_enstate;
	int *device_c;
	int c; 
	hipMalloc(&device_c, sizeof(int));
	amrenc <<<100, 500 >>>(device_c);
	start = clock();
	hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);
	finish = clock();
	duration = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("%f seconds,%d\n", duration,c);
	system("pause");
	//fclose(pcm_orig);
	//fclose(amrnb);
	//fclose(pcm_back);
	return 0;
}
